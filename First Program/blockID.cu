#include <hip/hip_runtime.h>


#include <stdio.h>

__global__ void printBlockDim() {
    printf("blockDim.x : %d, blockDim.y : %d, blockDim.z : %d\n", blockDim.x,
           blockDim.y, blockDim.z);
}

int main(int argc, char* argv[]) {
    int nx = 16, ny = 16, nz = 1;

    dim3 numThreads(8, 8, 1);
    dim3 numBlocks(nx / numThreads.x, ny / numThreads.y, nz / numThreads.z);

    printBlockDim<<<numBlocks, numThreads>>>();
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}
