
#include <hip/hip_runtime.h>
//#include <cuda_runtime.h>
//#include <device_launch_parameters.h>
//
//#include <stdio.h>
//
//__global__ void helloCUDAworld() {
//    printf("Hello CUDA world!\n");
//}
//
//int main(int argc, char* argv[]) {
//    int nx = 16, ny = 4, nz = 1;
//
//    dim3 numThreads(4, 1, 1);
//    dim3 numBlocks(nx / numThreads.x, ny / numThreads.y, nz / numThreads.z);
//
//    helloCUDAworld<<<numBlocks, numThreads>>>();
//    cudaDeviceSynchronize();
//    cudaDeviceReset();
//    return 0;
//}
