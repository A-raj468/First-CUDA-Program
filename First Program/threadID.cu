
#include <hip/hip_runtime.h>
//#include <cuda_runtime.h>
//#include <device_launch_parameters.h>
//
//#include <stdio.h>
//
//__global__ void printThreadID() {
//    printf("threadIdx.x : %d, threadIdx.y : %d, threadIdx.z : %d\n", threadIdx.x, threadIdx.y, threadIdx.z);
//}
//
// int main(int argc, char* argv[]) {
//     int nx = 16, ny = 16, nz = 1;
//
//     dim3 numThreads(8, 8, 1);
//     dim3 numBlocks(nx / numThreads.x, ny / numThreads.y, nz / numThreads.z);
//
//     printThreadID<<<numBlocks, numThreads>>>();
//     cudaDeviceSynchronize();
//     cudaDeviceReset();
//     return 0;
// }
